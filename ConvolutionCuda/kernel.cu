#include "hip/hip_runtime.h"


#include "hip/hip_runtime.h"
#include ""

// for syncthreads()
#ifdef __INTELLISENSE___
// in here put whatever is your favorite flavor of intellisense workarounds
void __syncthreads();
#endif
#include "hip/device_functions.h"




#include <stdio.h>


#define ARR_SIZE 7
#define MASK_SIZE 5



void printArr(float* arr) {
	for (int i = 0; i < ARR_SIZE; i++)
		printf("%4.0f  ", arr[i]);
	printf("\n");
}


/// 7.2 Simple Convolution 1D
//__global__ void convolution_1D_basic_kernel(float *N, float *M, float *P, int Mask_Width, int Width)
//{
//	int i = blockIdx.x * blockDim.x + threadIdx.x;
//
//	float Pvalue = 0.f;
//	int N_start_point = i - (Mask_Width / 2);
//	for (int j = 0; j < Mask_Width; j++) {
//		if (N_start_point + j >= 0 && N_start_point + j < Width) {
//			Pvalue += N[N_start_point + j] * M[j];
//		}
//	}
//	P[i] = Pvalue;
//}
//
//int main()
//{
//	float hstSrc[ARR_SIZE] = { 1.f, 2.f, 3.f, 4.f, 5.f, 6.f, 7.f };
//	float hstMask[MASK_SIZE] = { 3.f, 4.f, 5.f, 4.f, 3.f };
//	float hstResult[ARR_SIZE] = { 0.f };
//
//	float* devSrc = nullptr;
//	hipMalloc(&devSrc, sizeof(float) * ARR_SIZE);
//	hipMemcpy(devSrc, hstSrc, sizeof(float) * ARR_SIZE, hipMemcpyKind::hipMemcpyHostToDevice);
//
//	float* devMask = nullptr;
//	hipMalloc(&devMask, sizeof(float) * MASK_SIZE);
//	hipMemcpy(devMask, hstMask, sizeof(float) * MASK_SIZE, hipMemcpyKind::hipMemcpyHostToDevice);
//
//
//	float* devResult = nullptr;
//	hipMalloc(&devResult, sizeof(float) * ARR_SIZE);
//
//	convolution_1D_basic_kernel << < 1, ARR_SIZE >> > (devSrc, devMask, devResult, MASK_SIZE, ARR_SIZE);
//
//	hipMemcpy(hstResult, devResult, sizeof(float) * ARR_SIZE, hipMemcpyKind::hipMemcpyDeviceToHost);
//
//	printArr(hstResult);
//
//	return 0;
//}


/// 7.3 Convolution 1D with Contant Mask
//__constant__ float Mask[MASK_SIZE];
//
//__global__ void convolution_1D_const_kernel(float *N, float *P, int Mask_Width, int Width)
//{
//	int i = blockIdx.x * blockDim.x + threadIdx.x;
//
//	float Pvalue = 0.f;
//	int N_start_point = i - (Mask_Width / 2);
//	for (int j = 0; j < Mask_Width; j++) {
//		if (N_start_point + j >= 0 && N_start_point + j < Width) {
//			Pvalue += N[N_start_point + j] * Mask[j];
//		}
//	}
//	P[i] = Pvalue;
//}
//
//int main()
//{
//	float hstSrc[ARR_SIZE] = { 1.f, 2.f, 3.f, 4.f, 5.f, 6.f, 7.f };
//	float hstMask[MASK_SIZE] = { 3.f, 4.f, 5.f, 4.f, 3.f };
//	float hstResult[ARR_SIZE] = { 0.f };
//
//	float* devSrc = nullptr;
//	hipMalloc(&devSrc, sizeof(float) * ARR_SIZE);
//	hipMemcpy(devSrc, hstSrc, sizeof(float) * ARR_SIZE, hipMemcpyKind::hipMemcpyHostToDevice);
//
//	
//	hipMemcpyToSymbol(HIP_SYMBOL(Mask), hstMask, sizeof(float) * MASK_SIZE);
//
//
//	float* devResult = nullptr;
//	hipMalloc(&devResult, sizeof(float) * ARR_SIZE);
//
//	convolution_1D_const_kernel << < 1, ARR_SIZE >> > (devSrc, devResult, MASK_SIZE, ARR_SIZE);
//
//	hipMemcpy(hstResult, devResult, sizeof(float) * ARR_SIZE, hipMemcpyKind::hipMemcpyDeviceToHost);
//
//	printArr(hstResult);
//
//	return 0;
//}


/// 7.4 Convolution 1D with Halo cells
//__constant__ float Mask[MASK_SIZE];
//__global__ void convolution_1D_halo_kernel(float *N, float *P, int Mask_Width, int Width)
//{
//	int i = blockIdx.x * blockDim.x + threadIdx.x;
//	__shared__ float N_halo[ARR_SIZE + MASK_SIZE - 1];
//
//	int n = Mask_Width / 2;
//
//	int halo_index_left = (blockIdx.x - 1) * blockDim.x + threadIdx.x;
//	if (threadIdx.x >= blockDim.x - n) {
//		N_halo[threadIdx.x - (blockDim.x - n)]
//			= (halo_index_left < 0) ? 0 : N[halo_index_left];
//	}
//
//	N_halo[n + threadIdx.x] = N[i];
//
//	int halo_index_right = (blockIdx.x - 1) * blockDim.x + threadIdx.x;
//	if (threadIdx.x < n) {
//		N_halo[n + blockDim.x + threadIdx.x] =
//			(halo_index_right >= Width) ? 0 : N[halo_index_right];
//	}
//	__syncthreads();
//
//	float Pvalue = 0.f;
//	for (int j = 0; j < Mask_Width; j++) {
//		Pvalue += N_halo[threadIdx.x + j] * Mask[j];
//	}
//	P[i] = Pvalue;
//}
//
//int main()
//{
//	float hstSrc[ARR_SIZE] = { 1.f, 2.f, 3.f, 4.f, 5.f, 6.f, 7.f };
//	float hstMask[MASK_SIZE] = { 3.f, 4.f, 5.f, 4.f, 3.f };
//	float hstResult[ARR_SIZE] = { 0.f };
//
//	float* devSrc = nullptr;
//	hipMalloc(&devSrc, sizeof(float) * ARR_SIZE);
//	hipMemcpy(devSrc, hstSrc, sizeof(float) * ARR_SIZE, hipMemcpyKind::hipMemcpyHostToDevice);
//
//
//	hipMemcpyToSymbol(HIP_SYMBOL(Mask), hstMask, sizeof(float) * MASK_SIZE);
//
//
//	float* devResult = nullptr;
//	hipMalloc(&devResult, sizeof(float) * ARR_SIZE);
//
//	convolution_1D_halo_kernel << < 1, ARR_SIZE >> > (devSrc, devResult, MASK_SIZE, ARR_SIZE);
//
//	hipMemcpy(hstResult, devResult, sizeof(float) * ARR_SIZE, hipMemcpyKind::hipMemcpyDeviceToHost);
//
//	printArr(hstResult);
//
//	return 0;
//}


/// 7.5 Convolution 1D using general caching
//__constant__ float Mask[MASK_SIZE];
//__global__ void convolution_1D_caching_kernel(float *N, float *P, int Mask_Width, int Width)
//{
//	int i = blockIdx.x * blockDim.x + threadIdx.x;
//	__shared__ float N_sm[ARR_SIZE];
//
//	N_sm[threadIdx.x] = N[i];
//
//	__syncthreads();
//
//	int This_tile_start_point = blockIdx.x * blockDim.x;
//	int Next_tile_start_point = (blockIdx.x + 1) * blockDim.x;
//	int N_start_point = i - (Mask_Width / 2);
//
//	float Pvalue = 0.f;
//	for (int j = 0; j < Mask_Width; j++) {
//		int N_index = N_start_point + j;
//		if (N_index >= 0 && N_index < Width) {
//			if ((N_index >= This_tile_start_point)
//				&& (N_index < Next_tile_start_point)) {
//				Pvalue += N_sm[threadIdx.x + j - (Mask_Width / 2)] * Mask[j];
//			}
//			else {
//				Pvalue += N_sm[threadIdx.x + j] * Mask[j];
//			}
//		}
//	}
//	P[i] = Pvalue;
//}
//
//int main()
//{
//	float hstSrc[ARR_SIZE] = { 1.f, 2.f, 3.f, 4.f, 5.f, 6.f, 7.f };
//	float hstMask[MASK_SIZE] = { 3.f, 4.f, 5.f, 4.f, 3.f };
//	float hstResult[ARR_SIZE] = { 0.f };
//
//	float* devSrc = nullptr;
//	hipMalloc(&devSrc, sizeof(float) * ARR_SIZE);
//	hipMemcpy(devSrc, hstSrc, sizeof(float) * ARR_SIZE, hipMemcpyKind::hipMemcpyHostToDevice);
//
//
//	hipMemcpyToSymbol(HIP_SYMBOL(Mask), hstMask, sizeof(float) * MASK_SIZE);
//
//
//	float* devResult = nullptr;
//	hipMalloc(&devResult, sizeof(float) * ARR_SIZE);
//
//	convolution_1D_caching_kernel << < 1, ARR_SIZE >> > (devSrc, devResult, MASK_SIZE, ARR_SIZE);
//
//	hipMemcpy(hstResult, devResult, sizeof(float) * ARR_SIZE, hipMemcpyKind::hipMemcpyDeviceToHost);
//
//	printArr(hstResult);
//
//	return 0;
//}


/// 7.6 Convolution 2D with Halo cells
__constant__ float Mask[MASK_SIZE];
__global__ void convolution_1D_caching_kernel(float *N, float *P, int Mask_Width, int Width)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ float N_sm[ARR_SIZE];

	N_sm[threadIdx.x] = N[i];

	__syncthreads();

	int This_tile_start_point = blockIdx.x * blockDim.x;
	int Next_tile_start_point = (blockIdx.x + 1) * blockDim.x;
	int N_start_point = i - (Mask_Width / 2);

	float Pvalue = 0.f;
	for (int j = 0; j < Mask_Width; j++) {
		int N_index = N_start_point + j;
		if (N_index >= 0 && N_index < Width) {
			if ((N_index >= This_tile_start_point)
				&& (N_index < Next_tile_start_point)) {
				Pvalue += N_sm[threadIdx.x + j - (Mask_Width / 2)] * Mask[j];
			}
			else {
				Pvalue += N_sm[threadIdx.x + j] * Mask[j];
			}
		}
	}
	P[i] = Pvalue;
}

int main()
{
	float hstSrc[ARR_SIZE] = { 1.f, 2.f, 3.f, 4.f, 5.f, 6.f, 7.f };
	float hstMask[MASK_SIZE] = { 3.f, 4.f, 5.f, 4.f, 3.f };
	float hstResult[ARR_SIZE] = { 0.f };

	float* devSrc = nullptr;
	hipMalloc(&devSrc, sizeof(float) * ARR_SIZE);
	hipMemcpy(devSrc, hstSrc, sizeof(float) * ARR_SIZE, hipMemcpyKind::hipMemcpyHostToDevice);


	hipMemcpyToSymbol(HIP_SYMBOL(Mask), hstMask, sizeof(float) * MASK_SIZE);


	float* devResult = nullptr;
	hipMalloc(&devResult, sizeof(float) * ARR_SIZE);

	convolution_1D_caching_kernel << < 1, ARR_SIZE >> > (devSrc, devResult, MASK_SIZE, ARR_SIZE);

	hipMemcpy(hstResult, devResult, sizeof(float) * ARR_SIZE, hipMemcpyKind::hipMemcpyDeviceToHost);

	printArr(hstResult);

	return 0;
}
